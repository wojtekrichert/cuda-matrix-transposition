
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>

using namespace std;

int transponowanie(double tablica[1000][1000],double tab[1000][1000]){
  printf("przed: 1:%f ", tablica[0][1]);
  printf("2:%f \n", tablica[1][0]);
  for(int i=0; i<1000;i++){
    for(int j=0; j<1000;j++){
      tab[j][i]=tablica[i][j];
    }
  }
  printf("po: 1:%f ", tab[0][1]);
  printf("2:%f \n", tab[1][0]);
  return 1;
}

__global__ void transpose(double *matrix){
}


int main(void){
  clock_t begin = clock();
  static double tablica[1000][1000];
  static double tab[1000][1000];
  for(int i=0; i<1000;i++){
    for(int j=0; j<1000;j++){
      tablica[i][j]=i*1000+j+1;
    }
  }
  transponowanie(tablica,tab);
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("czas CPU: %f \n", elapsed_secs);

}
