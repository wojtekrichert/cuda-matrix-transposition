#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <iostream>
using namespace std;
const int size =1024;
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;

int zapelnianie(double ** matrix){
  for(int i = 0; i < size; ++i)
    for(int j = 0; j < size; ++j)
      matrix[i][j] = i*size+j;
  /*
  for(int i = 0; i < size; ++i)
    for(int j = 0; j < size; ++j)
      cout << matrix[i][j] << " ";
  */
  return 1;
}

int transponowanie(double ** matrix,double **CPUmatrix){
  clock_t begin = clock();
  for(int i=0; i<size;i++){
    for(int j=0; j<size;j++){
      CPUmatrix[j][i]=matrix[i][j];
    }
  }
  for( int x=0 ; x<size; x++){
    for(int y=0; y<size; y++){
      if(matrix[x][y]==CPUmatrix[x][y] && x!=y)
      printf("ups");
    }
  }
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("czas CPU: %f \n", elapsed_secs);
  return 1;
}

__global__ void transpose_global(double *d_array, double *d_GPUarray){
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS){
      d_GPUarray[x*width + (y+j)] = d_array[(y+j)*width + x];
    }
}

int main(void){
  dim3 dimGrid(size/TILE_DIM, size/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

  double ** matrix= new double *[size];
  double ** CPUmatrix= new double *[size];
  for(int i = 0; i < size; ++i){
    matrix[i]= new double[size];
    CPUmatrix[i]= new double[size];
  }
  zapelnianie(matrix);

  transponowanie(matrix,CPUmatrix);
  
  double *array,*d_array, *GPUarray, *d_GPUarray;
  array = (double*)malloc(size*size*sizeof(double));
  GPUarray = (double*)malloc(size*size*sizeof(double));
  int k=0;
  for(int x=0; x<size; ++x){
    for(int y=0;y<size;++y){
      array[k]=matrix[x][y];
      ++k;
    }
  }
  hipMalloc(&d_GPUarray, size*size*sizeof(double));
  hipMalloc(&d_array, size*size*sizeof(double));
  hipMemcpy(d_array, array, size*size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_GPUarray, GPUarray, size*size*sizeof(double), hipMemcpyHostToDevice);
  clock_t begin = clock();
  transpose_global<<<dimGrid, dimBlock>>>(d_array, d_GPUarray);
  for (int i = 0; i < NUM_REPS; i++){
    transpose_global<<<dimGrid, dimBlock>>>(d_array, d_GPUarray);
      hipDeviceSynchronize();
  }
  hipMemcpy(GPUarray, d_GPUarray,size*size*sizeof(double), hipMemcpyDeviceToHost);
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("\nczas GPU: %f \n", elapsed_secs);

hipFree(d_GPUarray);
hipFree(d_array);



}
