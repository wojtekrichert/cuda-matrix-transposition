#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <ctime>
#include <iostream>
using namespace std;
const int size =6144;
const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 100;

int zapelnianie(double ** matrix){
  for(int i = 0; i < size; ++i)
    for(int j = 0; j < size; ++j)
      matrix[i][j] = i*size+j;
  /*
  for(int i = 0; i < size; ++i)
    for(int j = 0; j < size; ++j)
      cout << matrix[i][j] << " ";
  */
  return 1;
}

int transponowanie(double ** matrix,double **CPUmatrix){
  for(int i=0; i<size;i++){
    for(int j=0; j<size;j++){
      CPUmatrix[j][i]=matrix[i][j];
    }
  }
  for( int x=0 ; x<size; x++){
    for(int y=0; y<size; y++){
      if(matrix[x][y]==CPUmatrix[x][y] && x!=y)
      printf("ups");
    }
  }
  return 1;
}

int sprawdzenie(double *GPUarray, double **CPUmatrix){
  int k=0;
  for(int x=0; x<size; ++x){
    for(int y=0;y<size;++y){
      if(GPUarray[k]!=CPUmatrix[x][y]){
        printf("ups");
      }
      ++k;
    }
  }
  return 1;
}

__global__ void transpose_global(double *d_array, double *d_GPUarray){
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;
    for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS){
      d_GPUarray[x*width + (y+j)] = d_array[(y+j)*width + x];
    }
}

__global__ void transpose_shared(double *d_array, double *d_GPUsharedarray){
  __shared__ double lol[TILE_DIM][TILE_DIM+1];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     lol[threadIdx.y+j][threadIdx.x] = d_array[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     d_GPUsharedarray[(y+j)*width + x] = lol[threadIdx.x][threadIdx.y + j];
}

int main(void){
  dim3 dimGrid(size/TILE_DIM, size/TILE_DIM, 1);
  dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);
  //transponowanie na CPU:-------------------------------
  clock_t begin1 = clock();
  double ** matrix= new double *[size];
  double ** CPUmatrix= new double *[size];
  for(int i = 0; i < size; ++i){
    matrix[i]= new double[size];
    CPUmatrix[i]= new double[size];
  }
  zapelnianie(matrix);
  transponowanie(matrix,CPUmatrix);
  clock_t end1 = clock();
  double elapsed_secs1 = 1+(double(end1 - begin1) / CLOCKS_PER_SEC);
  printf("czas CPU: %f \n", elapsed_secs1);
  //----------------------------------------------------



  double *array,*d_array, *GPUarray, *d_GPUarray, *d_GPUsharedarray, *GPUsharedarray;
  array = (double*)malloc(size*size*sizeof(double));
  GPUarray = (double*)malloc(size*size*sizeof(double));
  GPUsharedarray = (double*)malloc(size*size*sizeof(double));
  //splaszczenie 2d matrix na 1d array:----------------------
  int k=0;
  for(int x=0; x<size; ++x){
    for(int y=0;y<size;++y){
      array[k]=matrix[x][y];
      ++k;
    }
  }
  //---------------------------------------------------------
  hipMalloc(&d_GPUarray, size*size*sizeof(double));
  hipMalloc(&d_array, size*size*sizeof(double));
  hipMalloc(&d_GPUsharedarray, size*size*sizeof(double));
  hipMemcpy(d_array, array, size*size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_GPUarray, GPUarray, size*size*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_GPUsharedarray, GPUsharedarray, size*size*sizeof(double), hipMemcpyHostToDevice);

  //transpozycja z uzyciem pamieci globalnej:-------------------------
  clock_t begin = clock();
  transpose_global<<<dimGrid, dimBlock>>>(d_array, d_GPUarray);
  for (int i = 0; i < NUM_REPS; i++){
    transpose_global<<<dimGrid, dimBlock>>>(d_array, d_GPUarray);
  }
  hipMemcpy(GPUarray, d_GPUarray,size*size*sizeof(double), hipMemcpyDeviceToHost);
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("\nczas GPU global: %f \n", elapsed_secs);
  sprawdzenie(GPUarray, CPUmatrix);
  //-------------------------------------------------------------------

  //transpozycja z uzyciem pamieci shared : ---------------------
  clock_t begin2 = clock();
  transpose_shared<<<dimGrid, dimBlock>>>(d_array,d_GPUsharedarray);
  for (int a = 0; a < NUM_REPS; a++){
    transpose_shared<<<dimGrid, dimBlock>>>(d_array,d_GPUsharedarray);
  }

  hipMemcpy(GPUsharedarray, d_GPUsharedarray,size*size*sizeof(double), hipMemcpyDeviceToHost);
  clock_t end2 = clock();
  double elapsed_secs2 = double(end2 - begin2) / CLOCKS_PER_SEC;
  printf("\nczas GPU shared: %f \n", elapsed_secs2);
  sprawdzenie(GPUsharedarray, CPUmatrix);

  //-------------------------------------------------------------
  hipFree(d_GPUarray);
  hipFree(d_array);
  hipFree(d_GPUsharedarray);



}
