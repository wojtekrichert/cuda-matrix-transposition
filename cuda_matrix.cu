
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <ctime>

using namespace std;

int transponowanie(double tablica[1000][1000],double tab[1000][1000]){
  printf("przed: 1:%f ", tablica[0][1]);
  printf("2:%f \n", tablica[1][0]);
  for(int i=0; i<1000;i++){
    for(int j=0; j<1000;j++){
      tab[j][i]=tablica[i][j];
    }
  }
  return 1;
}

__global__ void transpose(double *matrix){
}


int main(void){
  clock_t begin = clock();
  static double tablica[1000][1000];
  static double tab[1000][1000];
  for(int i=0; i<1000;i++){
    for(int j=0; j<1000;j++){
      tablica[i][j]=i*1000+j+1;
    }
  }
  transponowanie(tablica,tab);
  for( int x=0 ; x<1000; x++){
    for(int y=0; y<1000; y++){
      if(tablica[x][y]==tab[x][y] && x!=y)
      printf("ups");
    }
  }
  printf("po: 1:%f ", tab[0][1]);
  printf("2:%f \n", tab[1][0]);
  clock_t end = clock();
  double elapsed_secs = double(end - begin) / CLOCKS_PER_SEC;
  printf("czas CPU: %f \n", elapsed_secs);

}
